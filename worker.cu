#include <hip/hip_runtime.h>

#include <stdio.h>  // For printf
#include <stdlib.h> // For exit

// Macro for checking CUDA errors
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in file '%s' in line %i : %s.\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// Function to check CUDA runtime version
void checkCudaVersion() {
    int driverVersion = 0;
    int runtimeVersion = 0;

    CUDA_CHECK(hipDriverGetVersion(&driverVersion));
    CUDA_CHECK(hipRuntimeGetVersion(&runtimeVersion));

    printf("CUDA Driver Version: %d\n", driverVersion);
    printf("CUDA Runtime Version: %d\n", runtimeVersion);
}

// CUDA kernel for matrix multiplication with shared memory
extern "C" __global__ void matrixMul(float* A, float* B, float* C, int N) {
    __shared__ float shared_A[16][16];
    __shared__ float shared_B[16][16];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float value = 0;

    for (int i = 0; i < (N + 15) / 16; ++i) {
        if (i * 16 + threadIdx.x < N && row < N)
            shared_A[threadIdx.y][threadIdx.x] = A[row * N + i * 16 + threadIdx.x];
        else
            shared_A[threadIdx.y][threadIdx.x] = 0.0;

        if (i * 16 + threadIdx.y < N && col < N)
            shared_B[threadIdx.y][threadIdx.x] = B[(i * 16 + threadIdx.y) * N + col];
        else
            shared_B[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        for (int j = 0; j < 16; ++j) {
            value += shared_A[threadIdx.y][j] * shared_B[j][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = value;
    }
}

// Function to run matrix multiplication on GPU
extern "C" void runMatrixMul(float* A, float* B, float* C, int N) {
    // Validate input
    if (N <= 0) {
        fprintf(stderr, "Invalid matrix size: %d. Size must be positive.\n", N);
        exit(EXIT_FAILURE);
    }

    size_t size = N * N * sizeof(float);

    // Check CUDA runtime version
    checkCudaVersion();

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void**)&d_A, size));
    CUDA_CHECK(hipMalloc((void**)&d_B, size));
    CUDA_CHECK(hipMalloc((void**)&d_C, size));

    // Check for memory allocation errors
    if (!d_A || !d_B || !d_C) {
        fprintf(stderr, "Memory allocation failed.\n");
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        exit(EXIT_FAILURE);
    }

    // Copy matrices from host to device
    CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

    // Define block and grid size
    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // Run kernel
    matrixMul<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA kernel launch error: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        exit(EXIT_FAILURE);
    }

    // Ensure kernel execution is complete
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    CUDA_CHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));

    // Check for memory copy errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA memory copy error: %s\n", hipGetErrorString(err));
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        exit(EXIT_FAILURE);
    }

    // Free device memory
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
}
